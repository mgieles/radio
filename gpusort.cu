#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

extern "C" void sort_func(float *r, int *jlist, int N)
{
  float *r_p;
  int *jlist_p;
  
  hipMalloc(&r_p    , N*sizeof(float));
  hipMalloc(&jlist_p, N*sizeof(int));
  
  hipMemcpy(r_p    ,  r    , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(jlist_p,  jlist, sizeof(int)*N, hipMemcpyHostToDevice); // Host -> Device
  
  thrust::device_ptr<float> r_d(r_p);
  thrust::device_ptr<int> jlist_d(jlist_p);
  
  thrust::sort_by_key(r_d, r_d + N, jlist_d);
  
  r_p     = thrust::raw_pointer_cast(r_d);
  jlist_p = thrust::raw_pointer_cast(jlist_d);
  
  hipMemcpy(r    , r_p, sizeof(float)*N, hipMemcpyDeviceToHost); // Device -> Host
  hipMemcpy(jlist, jlist_p, sizeof(int)*N, hipMemcpyDeviceToHost); // Device -> Host
  
  hipFree(r_p);
  hipFree(jlist_p);
}
