#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE 512

__global__ void gpu_phi(float *r, float *m, float *phi, int N)
{
  int i; 

  i = threadIdx.x + blockIdx.x*blockDim.x;
  
  if (i < N)
    {
      phi[i] = 0.0;
      for (int j=0; j<i; ++j)
	phi[i] -= m[j]/r[i];
      
      for (int j=i+1; j<N; ++j)
	phi[i] -= m[j]/r[j];
    }
}

extern "C" void getphi_func(float *r, float *m, float *phi, int N)
{
  float *r_d, *m_d, *phi_d;
  
  hipMalloc(&r_d   , sizeof(float)*N); 
  hipMalloc(&m_d   , sizeof(float)*N); 
  hipMalloc(&phi_d , sizeof(float)*N); 
  
  hipMemcpy(r_d  , r  , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(m_d  , m  , sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device
  hipMemcpy(phi_d, phi, sizeof(float)*N, hipMemcpyHostToDevice); // Host -> Device

  //  gpu_phi <<< 256, BLOCKSIZE >>>(r_d, m_d, phi_d, N);
  gpu_phi <<< ((N+BLOCKSIZE-1))/BLOCKSIZE,BLOCKSIZE >>>(r_d, m_d, phi_d, N);
  
  hipMemcpy(phi, phi_d, sizeof(float)*N, hipMemcpyDeviceToHost); // Device -> Host

  hipFree(r_d);
  hipFree(m_d);
  hipFree(phi_d);

  return;
}

